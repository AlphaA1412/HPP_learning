#include "hip/hip_runtime.h"
// MP 1
#include	<wb.h>

#define BLOCK_SIZE 256

int ceil(int a, int b){
    return int((a + b - 1) / b);
}
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < length){
        out[idx] = in1[idx] + in2[idx];
    }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here
    hipMalloc((void**) &device_input1, sizeof(float) * inputLength);
    hipMalloc((void**) &device_input2, sizeof(float) * inputLength);
    hipMalloc((void**) &device_output, sizeof(float) * inputLength);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here
    hipMemcpy(device_input1, hostInput1, sizeof(float) * length, hipMemcpyHostToDevice);
    hipMemcpy(device_input2, hostInput2, sizeof(float) * length, hipMemcpyHostToDevice);


    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    dim3 DimGrid(ceil(length / BLOCK_SIZE), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here
    vecAdd<<<DimGrid, DimBlock>>>(device_input1, device_input2, device_output);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here
    hipMemcpy(hostOutput, device_output, sizeof(float) * length, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here
    hipFree(device_input1);
    hipFree(device_input2);
    hipFree(device_output);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

