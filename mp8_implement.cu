#include "hip/hip_runtime.h"
#include	<wb.h>

#define SEGMENT_LENGTH 256
#define BLOCK_SIZE 256

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < len){
        out[idx] = in1[idx] + in2[idx];
    }
}

int min(int a, int b){
    if(a < b){
        return a;
    }
    return b;
}
int ceil(int a, int b){
    return (a + b - 1) / b;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    hipStream_t stream0, stream1, stream2, stream3;
    hipStreamCreate( &stream0);
    hipStreamCreate( &stream1);
    hipStreamCreate( &stream2);
    hipStreamCreate( &stream3);

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    // 1. Allocate memory on GPU
    hipMalloc((void**) &deviceInput1, sizeof(float) * 4 * SEGMENT_LENGTH);
    hipMalloc((void**) &deviceInput2, sizeof(float) * 4 * SEGMENT_LENGTH);
    hipMalloc((void**) &deviceOutput, sizeof(float) * 4 * SEGMENT_LENGTH);


    dim3 DimGrid(ceil(SEGMENT_LENGTH, BLOCK_SIZE), 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);

    // 2. do computation, Breadth First Kernel Issue
    for(int index = 0; index < inputLength; index += 4 * SEGMENT_LENGTH){
        int currentPtr1 = index;
        int currentPtr2 = currentPtr1 + SEGMENT_LENGTH;
        int currentPtr3 = currentPtr2 + SEGMENT_LENGTH;
        int currentPtr4 = currentPtr3 + SEGMENT_LENGTH;
        int length1 = 0, length2 = 0, length3 = 0, length4 = 0;
        
        // copy data
        if(currentPtr1 < inputLength){
            length1 = min(SEGMENT_LENGTH, inputLength - currentPtr1);
            cudaMemCpyAsync(&deviceInput1[0], hostInput1[currentPtr1], sizeof(float) * length1, hipMemcpyHostToDevice, stream0);
            cudaMemCpyAsync(&deviceInput2[0], hostInput2[currentPtr1], sizeof(float) * length1, hipMemcpyHostToDevice, stream0);
        }
        if(currentPtr2 < inputLength){
            length2 = min(SEGMENT_LENGTH, inputLength - currentPtr2);
            cudaMemCpyAsync(&deviceInput1[SEGMENT_LENGTH], hostInput1[currentPtr2], sizeof(float) * length2, hipMemcpyHostToDevice, stream1);
            cudaMemCpyAsync(&deviceInput2[SEGMENT_LENGTH], hostInput2[currentPtr2], sizeof(float) * length2, hipMemcpyHostToDevice, stream1);
        }
        if(currentPtr3 < inputLength){
            length3 = min(SEGMENT_LENGTH, inputLength - currentPtr3);
            cudaMemCpyAsync(&deviceInput1[SEGMENT_LENGTH * 2], hostInput1[currentPtr3], sizeof(float) * length3, hipMemcpyHostToDevice, stream2);
            cudaMemCpyAsync(&deviceInput2[SEGMENT_LENGTH * 2], hostInput2[currentPtr3], sizeof(float) * length3, hipMemcpyHostToDevice, stream2);
        }
        if(currentPtr4 < inputLength){
            length4 = min(SEGMENT_LENGTH, inputLength - currentPtr4);
            cudaMemCpyAsync(&deviceInput1[SEGMENT_LENGTH * 3], hostInput1[currentPtr4], sizeof(float) * length4, hipMemcpyHostToDevice, stream3);
            cudaMemCpyAsync(&deviceInput2[SEGMENT_LENGTH * 3], hostInput2[currentPtr4], sizeof(float) * length4, hipMemcpyHostToDevice, stream3);
        }
        // do calculation
        if(currentPtr1 < inputLength){
            vecAdd<<<DimGrid, DimBlock, stream0>>>(&deviceInput1[0], &deviceInput2[0], &deviceOutput[0], length1);
        }
        if(currentPtr2 < inputLength){
            vecAdd<<<DimGrid, DimBlock, stream1>>>(&deviceInput1[SEGMENT_LENGTH], &deviceInput2[SEGMENT_LENGTH], &deviceOutput[SEGMENT_LENGTH], length2);
        }
        if(currentPtr3 < inputLength){
            vecAdd<<<DimGrid, DimBlock, stream2>>>(&deviceInput1[SEGMENT_LENGTH * 2], &deviceInput2[SEGMENT_LENGTH * 2], &deviceOutput[SEGMENT_LENGTH * 2], length3);
        }
        if(currentPtr4 < inputLength){
            vecAdd<<<DimGrid, DimBlock, stream3>>>(&deviceInput1[SEGMENT_LENGTH * 3], &deviceInput2[SEGMENT_LENGTH * 3], &deviceOutput[SEGMENT_LENGTH * 3], length4);
        }


        // do memory copy from device to host
        if(currentPtr1 < inputLength){
            cudaMemCpyAsync(&hostOutput[currentPtr1], deviceOutput[0], sizeof(float) * length1, hipMemcpyDeviceToHost, stream0);
        }
        if(currentPtr2 < inputLength){
            cudaMemCpyAsync(&hostOutput[currentPtr2], deviceOutput[SEGMENT_LENGTH], sizeof(float) * length2, hipMemcpyDeviceToHost, stream1);
        }
        if(currentPtr3 < inputLength){
            cudaMemCpyAsync(&hostOutput[currentPtr3], deviceOutput[SEGMENT_LENGTH * 2], sizeof(float) * length3, hipMemcpyDeviceToHost, stream2);
        }
        if(currentPtr4 < inputLength){
            cudaMemCpyAsync(&hostOutput[currentPtr4], deviceOutput[SEGMENT_LENGTH * 3], sizeof(float) * length4, hipMemcpyDeviceToHost, stream3);
        }        
    }
    hipDeviceSynchronize();

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

