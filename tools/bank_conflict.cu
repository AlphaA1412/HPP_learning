#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void TestKernel(unsigned long long* time){
    __shared__ float shared_data[1024];
    unsigned long long startTime = clock();
    int tid = threadIdx.x;
    shared_data[tid * 2] ++ ;
    unsigned long long endTime = clock();
    time = endTime - startTime;
}


int main(){
    unsigned long long time;
    unsigned long long * dtime;
    hipMalloc((void**) &dtime, sizeof(unsigned long long));
    for(int index=0; index < 10; index++){
        TestKernel<<1, 32>>>(dtime);
        hipMemcpy(&time, dtime, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        std::cout <<"Time: "<<(time - 14) / 32 << std::endl;
        std::cout << std::endl;
    } 
    hipFree(dtime);
    hipDeviceReset();
    return 0;
}